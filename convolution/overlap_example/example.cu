
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hipfft/hipfft.h>

#define BLOCKSIZE 32
#define NUM_STREAMS 3

/**********/
/* iDivUp */
/*********/
int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/******************/
/* SUMMING KERNEL */
/******************/
__global__ void kernel(float2 *vec1, float2 *vec2, float2 *vec3, float2 *out, int N) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < N) {
        out[tid].x = vec1[tid].x + vec2[tid].x + vec3[tid].x;
        out[tid].y = vec1[tid].y + vec2[tid].y + vec3[tid].y;
    }

}


/********/
/* MAIN */
/********/
int main()
{
    const int N = 600000;
    const int Npartial = N / NUM_STREAMS;

    // --- Host input data initialization
    float2 *h_in1 = new float2[Npartial];
    float2 *h_in2 = new float2[Npartial];
    float2 *h_in3 = new float2[Npartial];
    for (int i = 0; i < Npartial; i++) {
        h_in1[i].x = 1.f;
        h_in1[i].y = 0.f;
        h_in2[i].x = 1.f;
        h_in2[i].y = 0.f;
        h_in3[i].x = 1.f;
        h_in3[i].y = 0.f;
    }

    // --- Host output data initialization
    float2 *h_out = new float2[N];

    // --- Registers host memory as page-locked (required for asynch cudaMemcpyAsync)
    gpuErrchk(hipHostRegister(h_in1, Npartial*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(h_in2, Npartial*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(h_in3, Npartial*sizeof(float2), hipHostRegisterPortable));

    // --- Device input data allocation
    float2 *d_in1;          gpuErrchk(hipMalloc((void**)&d_in1, N*sizeof(float2)));
    float2 *d_in2;          gpuErrchk(hipMalloc((void**)&d_in2, N*sizeof(float2)));
    float2 *d_in3;          gpuErrchk(hipMalloc((void**)&d_in3, N*sizeof(float2)));
    float2 *d_out1;         gpuErrchk(hipMalloc((void**)&d_out1, N*sizeof(float2)));
    float2 *d_out2;         gpuErrchk(hipMalloc((void**)&d_out2, N*sizeof(float2)));
    float2 *d_out3;         gpuErrchk(hipMalloc((void**)&d_out3, N*sizeof(float2)));
    float2 *d_out;          gpuErrchk(hipMalloc((void**)&d_out, N*sizeof(float2)));

    // --- Zero padding
    gpuErrchk(hipMemset(d_in1, 0, N*sizeof(float2)));
    gpuErrchk(hipMemset(d_in2, 0, N*sizeof(float2)));
    gpuErrchk(hipMemset(d_in3, 0, N*sizeof(float2)));

    // --- Creates CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) gpuErrchk(hipStreamCreate(&streams[i]));

    // --- Creates cuFFT plans and sets them in streams
    hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS);
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipfftPlan1d(&plans[i], N, HIPFFT_C2C, 1);
        hipfftSetStream(plans[i], streams[i]);
    }

    // --- Async memcopyes and computations
    gpuErrchk(hipMemcpyAsync(d_in1, h_in1, Npartial*sizeof(float2), hipMemcpyHostToDevice, streams[0]));
    gpuErrchk(hipMemcpyAsync(&d_in2[Npartial], h_in2, Npartial*sizeof(float2), hipMemcpyHostToDevice, streams[1]));
    gpuErrchk(hipMemcpyAsync(&d_in3[2*Npartial], h_in3, Npartial*sizeof(float2), hipMemcpyHostToDevice, streams[2]));
    hipfftExecC2C(plans[0], (hipfftComplex*)d_in1, (hipfftComplex*)d_out1, HIPFFT_FORWARD);
    hipfftExecC2C(plans[1], (hipfftComplex*)d_in2, (hipfftComplex*)d_out2, HIPFFT_FORWARD);
    hipfftExecC2C(plans[2], (hipfftComplex*)d_in3, (hipfftComplex*)d_out3, HIPFFT_FORWARD);

    for(int i = 0; i < NUM_STREAMS; i++) gpuErrchk(hipStreamSynchronize(streams[i]));

    kernel<<<iDivUp(BLOCKSIZE,N), BLOCKSIZE>>>(d_out1, d_out2, d_out3, d_out, N);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_out, d_out, N*sizeof(float2), hipMemcpyDeviceToHost));

    // for (int i=0; i<N; i++) printf("i = %i; real(h_out) = %f; imag(h_out) = %f\n", i, h_out[i].x, h_out[i].y);

    // --- Releases resources
    gpuErrchk(hipHostUnregister(h_in1));
    gpuErrchk(hipHostUnregister(h_in2));
    gpuErrchk(hipHostUnregister(h_in3));
    gpuErrchk(hipFree(d_in1));
    gpuErrchk(hipFree(d_in2));
    gpuErrchk(hipFree(d_in3));
    gpuErrchk(hipFree(d_out1));
    gpuErrchk(hipFree(d_out2));
    gpuErrchk(hipFree(d_out3));
    gpuErrchk(hipFree(d_out));

    for(int i = 0; i < NUM_STREAMS; i++) gpuErrchk(hipStreamDestroy(streams[i]));

    delete[] h_in1;
    delete[] h_in2;
    delete[] h_in3;
    delete[] h_out;

    hipDeviceReset();  

    return 0;
}