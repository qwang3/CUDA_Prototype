#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include "main.hpp"
#include "gpuErrchk.hpp"

__global__ void dot(hipfftDoubleComplex in_one[], hipfftDoubleComplex in_two[], hipfftDoubleComplex out[], int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        // some attempt at ILP ...
        float in_one_x = in_one[idx].x;
        float in_one_y = in_one[idx].y;
        float in_two_x = in_two[idx].x;
        float in_two_y = in_two[idx].y;

        out[idx].x = in_one_x * in_two_x;
        out[idx].y = in_one_y * in_two_y;

        // out[idx].x = in_one[idx].x * in_two[idx].x;
        // out[idx].y = in_one[idx].y * in_two[idx].y;
        idx += blockDim.x;
    }
}

double convolve_cufft(double input_array_1[], double input_array_2[], 
            double output_array[], int N_max, int batch) {
    hipfftDoubleComplex *in_one, *in_two, *tf_one, *tf_two, *out;
    hipfftHandle fft_plan;
    int dim[3] = {N_max, N_max, N_max};
    int N_total = pow(N_max, 3);
    int n_block = 1;
    int n_thread = 1024;

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    
    /* allocate memory */
    gpuErrchk(hipMalloc(&in_one, sizeof(hipfftDoubleComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&in_two, sizeof(hipfftDoubleComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&tf_one, sizeof(hipfftDoubleComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&tf_two, sizeof(hipfftDoubleComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&out, sizeof(hipfftDoubleComplex)*N_total*batch));

    /* copy over input */
    printf("[CUFFT] copying input \n");
    gpuErrchk(hipMemcpy2D((float*)in_one, 2*sizeof(float), 
                    input_array_1, sizeof(float), sizeof(float),
                    N_total*batch, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy2D((float*)in_two, 2*sizeof(float), 
                    input_array_2, sizeof(float), sizeof(float),
                    N_total*batch, hipMemcpyHostToDevice));
    hipfftPlanMany(&fft_plan, 3, dim, 
                    NULL, 0, 0,  
                    NULL, 0, 0, HIPFFT_C2C, batch);

    printf("[CUFFT] computing, timer on... \n");
    hipEventRecord(start);

    /* F = conv(f) */
    hipfftExecZ2Z(fft_plan, in_one, tf_one, HIPFFT_FORWARD);
    /* G = conv(g) */
    hipfftExecZ2Z(fft_plan, in_two, tf_two, HIPFFT_FORWARD);
    /* F dot G */
    dot<<<n_block, n_thread>>>(tf_one, tf_two, out, N_total*batch);
    /* conv^-1(F dot G) */
    hipfftExecZ2Z(fft_plan, out, out, HIPFFT_BACKWARD);

    hipEventRecord(stop);
    printf("[CUFFT] compute finished, timer off... \n");

    gpuErrchk(hipMemcpy2D(output_array, sizeof(float), 
                    out, 2*sizeof(float), sizeof(float),
                    N_total*batch, hipMemcpyDeviceToHost));

    gpuErrchk(hipEventSynchronize(stop));

    hipfftDestroy(fft_plan);
    gpuErrchk(hipFree(in_one)); 
    gpuErrchk(hipFree(in_two));
    gpuErrchk(hipFree(tf_one)); 
    gpuErrchk(hipFree(tf_two));
    gpuErrchk(hipFree(out));

    // miliseconds
    float cufft_duration;
    hipEventElapsedTime(&cufft_duration, start, stop); 
    return cufft_duration*1e-3;
}
