#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include "main.hpp"
#include "gpuErrchk.hpp"

static __global__ void point_mul(hipfftComplex *in_one, const hipfftComplex *in_two,  
                                 int N, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        hipfftComplex in_1 = in_one[idx];
        hipfftComplex in_2 = in_two[idx];
        hipfftComplex out;

        out.x = (in_1.x * in_2.x - in_1.y * in_2.y)*scale;
        out.y = (in_1.x * in_2.y + in_1.y * in_2.x)*scale;
        in_one[idx] = out;
        // printf("    threadIdx %d: out = %f out = %f \n", idx, in_one[idx].x, in_one[idx].y);

        idx += blockDim.x*gridDim.x;
    }
}

static __global__ void print_mem(hipfftComplex * mem, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=idx; i<N; i+= blockDim.x*gridDim.x) {
        if (mem[i].x != 0) {
            printf("    threadIdx %d: mem[%d].x= %f .y = %f\n", i, i, mem[i].x, mem[i].y);
        }
    }
}

float convolve_cufft_1(Complex input_array_1[], Complex input_array_2[], 
            Complex output_array[], int N_max, int batch) {
    hipfftComplex *in_one, *in_two;
    hipfftHandle fft_plan;
    int dim[3] = {N_max, N_max, N_max};
    int N_total = pow(N_max, 3);
    // int n_block = 32;
    // int n_thread = 1024;

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    
    /* allocate memory */
    gpuErrchk(hipMalloc(&in_one, sizeof(hipfftComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&in_two, sizeof(hipfftComplex)*N_total*batch));

    int minGridSize, gridSize, blockSize;
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, point_mul, 0, N_total*batch));
    gridSize = (N_total*batch + blockSize - 1) / blockSize; 

    hipfftPlanMany(&fft_plan, 3, dim, 
        NULL, 1, N_total,  
        NULL, 1, N_total, HIPFFT_C2C, batch);

    /* for debugging */


    /* copy over input */
    hipEventRecord(start);
    gpuErrchk(hipMemcpy(in_one, input_array_1, sizeof(Complex)*N_total*batch, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(in_two, input_array_2, sizeof(Complex)*N_total*batch, hipMemcpyHostToDevice));


    /* F = conv(f) */
    hipfftExecC2C(fft_plan, in_one, in_one, HIPFFT_FORWARD);
    /* G = conv(g) */
    hipfftExecC2C(fft_plan, in_two, in_two, HIPFFT_FORWARD);

    /* F dot G */
    // hipDeviceSynchronize();
    point_mul<<<gridSize, blockSize>>>(in_one, in_two, N_total*batch, 1.0f/N_total);
    // hipDeviceSynchronize();
    /* conv^-1(F dot G) */
    // printf("### [CUDA] Printing out before fft ..\n");
    // print_mem<<<gridSize, blockSize>>>(in_one, N_total*batch);
    // hipDeviceSynchronize();

    hipfftExecC2C(fft_plan, in_one, in_one, HIPFFT_BACKWARD);
    // hipDeviceSynchronize();

    // printf("### [CUDA] Printing out after fft..\n");
    // print_mem<<<gridSize, blockSize>>>(out, N_total*batch);

    gpuErrchk(hipMemcpy(output_array, in_one, sizeof(Complex)*N_total*batch, hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    gpuErrchk(hipEventSynchronize(stop));

    hipfftDestroy(fft_plan);
    gpuErrchk(hipFree(in_one)); 
    gpuErrchk(hipFree(in_two));

    // miliseconds
    float cufft_duration;
    hipEventElapsedTime(&cufft_duration, start, stop); 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return cufft_duration*1e-3;
}
