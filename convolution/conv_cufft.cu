#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "main.hpp"

__global__ void dot(hipfftComplex in_one[], hipfftComplex in_two[], hipfftComplex out[], int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        out[idx].x = in_one[idx].x * in_two[idx].x;
        out[idx].y = in_one[idx].y * in_two[idx].y;
        idx += blockDim.x;
    }
}

double convolve_cufft(float input_array_1[], float input_array_2[], 
            float output_array[], int N_max) {
    hipfftComplex *in_one, *in_two, *tf_one, *tf_two, *out;
    hipfftHandle fft_plan;
    int N_total = pow(N_max, 3);
    int n_block = 1;
    int n_thread = 1024;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    /* allocate memory */
    hipMalloc(&in_one, sizeof(hipfftComplex)*N_total);
    hipMalloc(&in_two, sizeof(hipfftComplex)*N_total);
    hipMalloc(&tf_one, sizeof(hipfftComplex)*N_total);
    hipMalloc(&tf_two, sizeof(hipfftComplex)*N_total);
    hipMalloc(&out, sizeof(hipfftComplex)*N_total);

    /* copy over input */
    printf("[CUFFT] copying input \n");
    hipMemcpy2D((float*)in_one, 2*sizeof(float), 
                    input_array_1, sizeof(float), sizeof(float),
                N_total, hipMemcpyHostToDevice);
    hipMemcpy2D((float*)in_two, 2*sizeof(float), 
                    input_array_2, sizeof(float), sizeof(float),
                N_total, hipMemcpyHostToDevice);

    printf("[CUFFT] computing, timer on... \n");
    hipEventRecord(start);
    hipfftPlan3d(&fft_plan, N_max, N_max, N_max, HIPFFT_C2C);
    /* F = conv(f) */
    hipfftExecC2C(fft_plan, in_one, tf_one, HIPFFT_FORWARD);
    /* G = conv(g) */
    hipfftExecC2C(fft_plan, in_two, tf_two, HIPFFT_FORWARD);
    /* F dot G */
    dot<<<n_block, n_thread>>>(tf_one, tf_two, out, N_total);
    /* conv^-1(F dot G) */
    hipfftExecC2C(fft_plan, out, out, HIPFFT_BACKWARD);

    hipEventRecord(stop);
    printf("[CUFFT] compute finished, timer off... \n");

    hipMemcpy2D(output_array, sizeof(float), 
                    out, 2*sizeof(float), sizeof(float),
                N_total, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    hipfftDestroy(fft_plan);
    hipFree(in_one); 
    hipFree(in_two);
    hipFree(tf_one); 
    hipFree(tf_two);
    hipFree(out);

    // miliseconds
    float cufft_duration;
    hipEventElapsedTime(&cufft_duration, start, stop); 
    return cufft_duration*1e-3;
}
