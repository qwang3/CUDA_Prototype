#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include "main.hpp"
#include "gpuErrchk.hpp"

__global__ void dot(hipfftComplex in_one[], hipfftComplex in_two[], hipfftComplex out[], int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        // some attempt at ILP ...
        float in_one_x = in_one[idx].x;
        float in_one_y = in_one[idx].y;
        float in_two_x = in_two[idx].x;
        float in_two_y = in_two[idx].y;

        out[idx].x = in_one_x * in_two_x;
        out[idx].y = in_one_y * in_two_y;

        // out[idx].x = in_one[idx].x * in_two[idx].x;
        // out[idx].y = in_one[idx].y * in_two[idx].y;
        idx += blockDim.x;
    }
}

double convolve_cufft(float input_array_1[], float input_array_2[], 
            float output_array[], int N_max, int batch) {
    hipfftComplex *in_one, *in_two, *tf_one, *tf_two, *out;
    hipfftHandle fft_plan;
    int dim[3] = {N_max, N_max, N_max};
    int N_total = pow(N_max, 3);
    int n_block = 1;
    int n_thread = 1024;

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    
    /* allocate memory */
    gpuErrchk(hipMalloc(&in_one, sizeof(hipfftComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&in_two, sizeof(hipfftComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&tf_one, sizeof(hipfftComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&tf_two, sizeof(hipfftComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&out, sizeof(hipfftComplex)*N_total*batch));

    /* copy over input */
    printf("[CUFFT] copying input \n");
    gpuErrchk(hipMemcpy2D((float*)in_one, 2*sizeof(float), 
                    input_array_1, sizeof(float), sizeof(float),
                    N_total*batch, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy2D((float*)in_two, 2*sizeof(float), 
                    input_array_2, sizeof(float), sizeof(float),
                    N_total*batch, hipMemcpyHostToDevice));
    hipfftPlanMany(&fft_plan, 3, dim, 
                    NULL, 0, 0,  
                    NULL, 0, 0, HIPFFT_C2C, batch);

    printf("[CUFFT] computing, timer on... \n");
    hipEventRecord(start);

    /* F = conv(f) */
    hipfftExecC2C(fft_plan, in_one, tf_one, HIPFFT_FORWARD);
    /* G = conv(g) */
    hipfftExecC2C(fft_plan, in_two, tf_two, HIPFFT_FORWARD);
    /* F dot G */
    dot<<<n_block, n_thread>>>(tf_one, tf_two, out, N_total*batch);
    /* conv^-1(F dot G) */
    hipfftExecC2C(fft_plan, out, out, HIPFFT_BACKWARD);

    hipEventRecord(stop);
    printf("[CUFFT] compute finished, timer off... \n");

    gpuErrchk(hipMemcpy2D(output_array, sizeof(float), 
                    out, 2*sizeof(float), sizeof(float),
                    N_total*batch, hipMemcpyDeviceToHost));

    gpuErrchk(hipEventSynchronize(stop));

    hipfftDestroy(fft_plan);
    gpuErrchk(hipFree(in_one)); 
    gpuErrchk(hipFree(in_two));
    gpuErrchk(hipFree(tf_one)); 
    gpuErrchk(hipFree(tf_two));
    gpuErrchk(hipFree(out));

    // miliseconds
    float cufft_duration;
    hipEventElapsedTime(&cufft_duration, start, stop); 
    return cufft_duration*1e-3;
}
