#include "hip/hip_runtime.h"
#include <hipfft/hipfftw.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "main.hpp"
#include "gpuErrchk.hpp"

static __global__ void point_mul(hipfftComplex *in_one, const hipfftComplex *in_two, 
                                int N, int scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        hipfftComplex in_1 = in_one[idx];
        hipfftComplex in_2 = in_two[idx];
        hipfftComplex out;

        out.x = (in_1.x * in_2.x - in_1.y * in_2.y)/scale;
        out.y = (in_1.x * in_2.y + in_1.y * in_2.x)/scale;
        in_one[idx] = out;
        idx += blockDim.x*gridDim.x;
    }
}

float convolve_cufft_3(Complex input_array_1[], Complex input_array_2[], 
            Complex output_array[], int N_max, int batch) {
    hipfftComplex *in_one, *in_two;
    int N_total = pow(N_max, 3);

    // Registers host memory as page-locked (required for asynch hipMemcpyAsync)
    gpuErrchk(hipHostRegister(input_array_1, N_total*batch*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(input_array_2, N_total*batch*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(output_array, N_total*batch*sizeof(float2), hipHostRegisterPortable));

    // Calculate optimal block and grid sizes
    int minGridSize, gridSize, blockSize;
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, point_mul, 0, N_total*batch));
    gridSize = (N_total*batch + blockSize - 1) / blockSize; 

    // streams 
    const int num_streams = batch*2; 
    hipStream_t stream[num_streams];
    hipfftHandle *fft_plan = new hipfftHandle[num_streams];
    for (int i=0; i<num_streams; i++) {
        gpuErrchk(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));
        hipfftPlan3d(&fft_plan[i], N_max, N_max, N_max, HIPFFT_C2C);
        hipfftSetStream(fft_plan[i], stream[i]);
    }

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    
    /* allocate memory */
    gpuErrchk(hipMalloc(&in_one, sizeof(hipfftComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&in_two, sizeof(hipfftComplex)*N_total*batch));

    hipProfilerStart();
    hipEventRecord(start);
    // #pragma unroll
    for (int i=0; i<batch; i++) {
        int loc = i * N_total;
        gpuErrchk(hipMemcpyAsync(&in_one[loc], &input_array_1[loc], sizeof(float2)*N_total, 
                                hipMemcpyHostToDevice, stream[2*i]));
        hipfftExecC2C(fft_plan[2*i], &in_one[loc], &in_one[loc], HIPFFT_FORWARD);
        gpuErrchk(hipMemcpyAsync(&in_two[loc], &input_array_2[loc], sizeof(float2)*N_total, 
                                hipMemcpyHostToDevice, stream[2*i+1]));
        hipfftExecC2C(fft_plan[2*i+1], &in_two[loc], &in_two[loc], HIPFFT_FORWARD);
        gpuErrchk(hipStreamSynchronize(stream[2*i+1]));
        point_mul<<<gridSize, blockSize, 0, stream[2*i]>>>(&in_one[i], &in_two[i], N_total, N_total);
        hipfftExecC2C(fft_plan[2*i], &in_one[loc], &in_one[loc], HIPFFT_BACKWARD);
        gpuErrchk(hipMemcpyAsync(&output_array[loc], &in_one[loc], sizeof(float2)*N_total, 
                                hipMemcpyDeviceToHost, stream[2*i]));
    }

    hipEventRecord(stop);
    gpuErrchk(hipEventSynchronize(stop));
    hipProfilerStop();

    gpuErrchk(hipFree(in_one)); 
    gpuErrchk(hipFree(in_two));

    for (int i=0; i<num_streams; i++) {
        gpuErrchk(hipStreamDestroy(stream[i]));
        hipfftDestroy(fft_plan[i]);
    }
    delete[] fft_plan;

    gpuErrchk(hipHostUnregister(input_array_1));
    gpuErrchk(hipHostUnregister(input_array_2));
    gpuErrchk(hipHostUnregister(output_array));


    // miliseconds
    float cufft_duration;
    hipEventElapsedTime(&cufft_duration, start, stop); 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return cufft_duration*1e-3;
}


float convolve_cufft_2(Complex input_array_1[], Complex input_array_2[], 
    Complex output_array[], int N_max, int batch) {
    hipfftComplex *in_one, *in_two;
    int N_total = pow(N_max, 3);

    // Registers host memory as page-locked (required for asynch hipMemcpyAsync)
    gpuErrchk(hipHostRegister(input_array_1, N_total*batch*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(input_array_2, N_total*batch*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(output_array, N_total*batch*sizeof(float2), hipHostRegisterPortable));

    // Calculate optimal block and grid sizes
    int minGridSize, gridSize, blockSize;
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, point_mul, 0, N_total*batch));
    gridSize = (N_total*batch + blockSize - 1) / blockSize; 

    // streams 
    const int num_streams = batch; 
    hipStream_t stream[num_streams];
    hipfftHandle *fft_plan = new hipfftHandle[num_streams];
    for (int i=0; i<num_streams; i++) {
        gpuErrchk(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));
        hipfftPlan3d(&fft_plan[i], N_max, N_max, N_max, HIPFFT_C2C);
        hipfftSetStream(fft_plan[i], stream[i]);
    }

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    /* allocate memory */
    gpuErrchk(hipMalloc(&in_one, sizeof(hipfftComplex)*N_total*batch));
    gpuErrchk(hipMalloc(&in_two, sizeof(hipfftComplex)*N_total*batch));

    hipEventRecord(start);
    for (int i=0; i<num_streams; i++) {
        int loc = i * N_total;
        gpuErrchk(hipMemcpyAsync(&in_one[loc], &input_array_1[loc], sizeof(float2)*N_total, 
                                hipMemcpyHostToDevice, stream[i]));
        hipfftExecC2C(fft_plan[i], &in_one[loc], &in_one[loc], HIPFFT_FORWARD);
    }
    for (int i=0; i<num_streams; i++) {
        int loc = i * N_total;
        gpuErrchk(hipMemcpyAsync(&in_two[loc], &input_array_2[loc], sizeof(float2)*N_total, 
                                hipMemcpyHostToDevice, stream[i]));
        hipfftExecC2C(fft_plan[i], &in_two[loc], &in_two[loc], HIPFFT_FORWARD);
    }
    for (int i=0; i<num_streams; i++) {
        gpuErrchk(hipStreamSynchronize(stream[i]));
    }
    point_mul<<<gridSize, blockSize, 0, stream[0]>>>(in_one, in_two, N_total*batch, N_total);
    for (int i=0; i<num_streams; i++) {
        int loc = i * N_total;
        hipfftExecC2C(fft_plan[i], &in_one[loc], &in_one[loc], HIPFFT_BACKWARD);
        gpuErrchk(hipMemcpyAsync(&output_array[loc], &in_one[loc], sizeof(float2)*N_total, 
                            hipMemcpyDeviceToHost, stream[i]));
    }
    hipEventRecord(stop);
    gpuErrchk(hipEventSynchronize(stop));

    gpuErrchk(hipFree(in_one)); 
    gpuErrchk(hipFree(in_two));
    for (int i=0; i<num_streams; i++) {
        gpuErrchk(hipStreamDestroy(stream[i]));
        hipfftDestroy(fft_plan[i]);
    }
    delete[] fft_plan;

    gpuErrchk(hipHostUnregister(input_array_1));
    gpuErrchk(hipHostUnregister(input_array_2));
    gpuErrchk(hipHostUnregister(output_array));


    // miliseconds
    float cufft_duration;
    hipEventElapsedTime(&cufft_duration, start, stop); 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return cufft_duration*1e-3;
}

