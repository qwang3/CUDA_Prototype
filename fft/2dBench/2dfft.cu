#include "hip/hip_runtime.h"
#include "2dfft.hpp"

using namespace std;

/* Calcuate FFT with cuFTT */

float fft_cuda(double** idata, double** odata, int Nx, int Ny) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Allocate memory for data on device, then copy data */
    double *idata_c, *odata_c;
    hipMalloc(&odata_c, sizeof(double) * Nx * Ny);
    hipMalloc(&idata_c, sizeof(double) * Nx * Ny);

    hipfftDoubleComplex *idata_cx, *odata_cx;
    hipMalloc(&idata_cx, sizeof(hipfftDoubleComplex) * Nx * Ny);
    hipMalloc(&odata_cx, sizeof(hipfftDoubleComplex) * Nx * Ny);

    for (int i=0; i<Nx; i++) {
        hipMemcpy(&idata_c[i*Ny], &idata[i][0], sizeof(double)*Ny, hipMemcpyHostToDevice);
    }

    /* Convert data into hipfftDoubleComplex */
    /* set 1 block with 256 threads */
    real2complex<<<1, 8>>>(idata_c, idata_cx, Nx*Ny);
    hipDeviceSynchronize();
    /* FFT Plans */
    hipfftHandle plan;
    hipfftPlan2d(&plan, Nx, Ny, HIPFFT_Z2Z);

    // for (int i=0; i<Nx; i++) {
    //     for (int j=0; j<Ny; j++) {
    //         printf("%d", i*Nx+j);
    //         printf("%f", idata_c[i*Nx+j]);
    //     }
    //     printf("\n");
    // }

    // auto start = chrono::high_resolution_clock::now();
    /* Forward FFT */
    hipEventRecord(start);
    hipfftExecZ2Z(plan, idata_cx, odata_cx, HIPFFT_FORWARD);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    /* stop the time */
    /* std::chrono::_V2::system_clock::time_point finish */ 
    float duration = 0; // milliseconds
    hipEventElapsedTime(&duration, start, stop);
    /* Convert cufft back to double array */
    /* set 1 block with 8 threads */

    complex2real<<<1, 8>>>(odata_cx, odata_c, Nx*Ny);
    hipDeviceSynchronize();

    for (int i=0; i<Nx; i++) {
        hipMemcpy(&odata[i][0], &odata_c[i*Ny], sizeof(double)*Ny, hipMemcpyDeviceToHost);
    }

    hipfftDestroy(plan);
    hipFree(idata_c);
    hipFree(odata_c);
    hipFree(idata_cx);
    hipFree(odata_cx);

    return duration;
}

/* convert a double array to cuffComplex data type. Imaginary parts are
 * set to 0 
 */
__global__ void real2complex(double *f, hipfftDoubleComplex *fc, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < N) {
        fc[index].x = f[index];
        fc[index].y = 0;
        index += stride;
    }
}

/* convert a cuffComplex data type to a double array.
 */
 __global__ void complex2real(hipfftDoubleComplex *fc, double *f, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    while (index < N) {
        f[index] = fc[index].x;
        index += stride;
        
    }
}