#include "hip/hip_runtime.h"
#include <ctime>
#include <chrono>
#include <stdio.h>
#include <iostream>
#include <cmath>

#include <fftw3.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

using namespace std;

__global__ void real2complex(double *f, hipfftComplex *fc, int N);
__global__ void complex2real(hipfftComplex *fc, double *f, int N);

void usage(int argc, char **argv) {
    cout << "usage: " << argv[0] << " Nx batch" << endl;
    cout << "Nx (int): number of element in array" << endl;
    cout << "batch (int): number of arrys to compute" << endl;
    
}

int main(int argc, char **argv) {

    if (argc != 3) {
        usage(argc, argv);
        exit(1);
    }
    int Nx = atoi(argv[1]);
    int batch = atoi(argv[2]);


    /* randomly generate data */
    double *idata = new double[Nx*batch];
    double *odata = new double[Nx*batch];
    for (int i=0; i<Nx*batch; i++) {
        idata[i] = rand() / (double)RAND_MAX;
    }

    /* Allocate memory for data on device, then copy data */
    cout << "Allocating memory for data on device" << endl;
    double *idata_c;
    hipfftComplex *idata_cx;
    double *odata_c; 
    hipfftComplex *odata_cx;
    hipMalloc(&odata_c, sizeof(double) * Nx*batch);
    hipMalloc(&idata_c, sizeof(double) * Nx*batch);
    hipMalloc(&idata_cx, sizeof(hipfftComplex) * Nx*batch);
    hipMalloc(&odata_cx, sizeof(hipfftComplex) * Nx*batch);

    hipMemcpy(idata_c, idata, sizeof(double) * Nx*batch, hipMemcpyHostToDevice);

    /* FFT Plans */
    hipfftHandle plan;
    hipfftPlan1d(&plan, Nx, HIPFFT_C2C, batch);

    /* start the time now */
    auto start = chrono::high_resolution_clock::now();

    /* Convert data into hipfftComplex */
    /* set 1 block with 128 threads */
    cout << "converting real2complex" << endl;
    real2complex<<<1, 128>>>(idata_c, idata_cx, Nx*batch);
    hipDeviceSynchronize();

    /* Forward FFT */
    cout << "Forward FFT" << endl;
    hipfftExecC2C(plan, idata_cx, odata_cx, HIPFFT_FORWARD);

    /* Inverse FFT */
    cout << "Inverse FFT" << endl;
    hipfftExecC2C(plan, odata_cx, idata_cx, HIPFFT_BACKWARD);

    /* Convert cufft back to double array */
    /* set 1 block with 128 threads */

    cout << "converting complex2real" << endl;
    complex2real<<<1, 128>>>(idata_cx, odata_c, Nx*batch);
    hipDeviceSynchronize();

    hipMemcpy(odata, odata_c, sizeof(double)*Nx*batch, hipMemcpyDeviceToHost);

    /* stop the time */
    auto finish = chrono::high_resolution_clock::now();

    for (int i = 0; i < Nx*batch; i++){
        if (abs(idata[i] - odata[i]) > 1e-6) {
            cout << "[ERROR] Mismatch at " << i;
            cout << " idata[" << i << "]= " << idata[i];
            cout << " odata[" << i << "]= " << odata[i] << endl;
        }
    }

    auto duration = chrono::duration_cast<chrono::milliseconds>(finish - start);
    cout << "Calculation ran for " << duration.count() << "ms" << endl;

    hipfftDestroy(plan);
    free(odata);
    free(idata);
    hipFree(idata_c);
    hipFree(idata_cx);
    hipFree(odata_c);
}

/* convert a double array to cuffComplex data type. Imaginary parts are
 * set to 0 
 */
__global__ void real2complex(double *f, hipfftComplex *fc, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < N) {
        fc[index].x = f[index];
        fc[index].y = 0;
        index += stride;
    }
}

/* convert a cuffComplex data type to a double array.
 */
 __global__ void complex2real(hipfftComplex *fc, double *f, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < N) {
        f[index] = fc[index].x / N;
        index += stride;
    }
}