#include "hip/hip_runtime.h"
#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <cmath>

#include <fftw3.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

using namespace std; 

__global__ void real2complex(double *f, hipfftComplex *fc, int N);
__global__ void complex2real(hipfftComplex *fc, double *f, int N);

void usage(int argc, char **argv) {
    cout << "usage: " << argv[0] << " Nx Ny" << endl;
    cout << "Nx (int): number of elemenst in x direction" << endl;
    cout << "Nx (int): number of elemenst in y direction" << endl;
}

int main(int argc, char **argv) {

    if (argc != 3) {
        usage(argc, argv);
        exit(1);
    }
    int Nx = atoi(argv[1]);
    int Ny = atoi(argv[2]);

    /* randomly generate data */
    double *idata = new double[Nx*Ny];
    double *odata = new double[Nx*Ny];
    for (int i=0; i<Nx*Ny; i++) {
        idata[i] = rand() / (double) RAND_MAX;
    }

    /* Allocate memory for data on device, then copy data */
    cout << "Allocating memory for data on device" << endl;
    double *idata_c;
    hipfftComplex *idata_cx;
    double *odata_c; 
    hipfftComplex *odata_cx;
    hipMalloc(&odata_c, sizeof(double) * Nx*Ny);
    hipMalloc(&idata_c, sizeof(double) * Nx*Ny);
    hipMalloc(&idata_cx, sizeof(hipfftComplex) * Nx*Ny);
    hipMalloc(&odata_cx, sizeof(hipfftComplex) * Nx*Ny);

    hipMemcpy(idata_c, idata, sizeof(double) * Nx*Ny, hipMemcpyHostToDevice);

    /* Convert data into hipfftComplex */
    /* set 1 block with 256 threads */
    cout << "converting real2complex" << endl;
    real2complex<<<2, 128>>>(idata_c, idata_cx, Nx*Ny);

    /* FFT Plans */
    int n[2] = {Nx, Ny};
    hipfftHandle plan;
    hipfftPlanMany(&plan, 2, n,
                    NULL, 1, 0, 
                    NULL, 1, 0, 
                    HIPFFT_C2C, 1);

    /* Forward FFT */
    hipfftExecC2C(plan, idata_cx, odata_cx, HIPFFT_FORWARD);

    /* Inverse FFT */
    hipfftExecC2C(plan, odata_cx, idata_cx, HIPFFT_BACKWARD);

    /* Convert cufft back to double array */
    /* set 1 block with 256 threads */

    cout << "converting complex2real" << endl;
    complex2real<<<2, 128>>>(idata_cx, odata_c, Nx);

    hipMemcpy(odata, odata_c, sizeof(double)*Nx, hipMemcpyDeviceToHost);

    /* Normalize result */
    for (int i=0; i < Nx*Ny; i++) {
        odata[i] /= Nx*Ny;
    }


    for (int i = 0; i < Nx; i++){
        if (abs(idata[i] - odata[i]) > 1e-6) {
            cout << "[ERROR] Mismatch at " << i;
            cout << " idata[" << i << "]= " << idata[i];
            cout << " odata[" << i << "]= " << odata[i] << endl;
        }
    }

    hipfftDestroy(plan);
    free(odata);
    free(idata);
    hipFree(idata_c);
    hipFree(idata_cx);
    hipFree(odata_c);
}

/* convert a double array to cuffComplex data type. Imaginary parts are
 * set to 0 
 */
__global__ void real2complex(double *f, hipfftComplex *fc, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < N) {
        fc[index].x = f[index];
        fc[index].y = 0;
        index += stride;
    }
}

/* convert a cuffComplex data type to a double array.
 */
 __global__ void complex2real(hipfftComplex *fc, double *f, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < N) {
        f[index] = fc[index].x;
        index += stride;
    }
}