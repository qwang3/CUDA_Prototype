#include "hip/hip_runtime.h"
#include "1dfft.hpp"

using namespace std;

/* Calcuate FFT with cuFTT */

float fft_cuda(const double* idata, double* odata, int Nx) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    /* Allocate memory for data on device, then copy data */
    double *idata_c, *odata_c;
    hipfftDoubleComplex *idata_cx, *odata_cx;
    hipMalloc(&odata_c, sizeof(double) * Nx);
    hipMalloc(&idata_c, sizeof(double) * Nx);
    hipMalloc(&idata_cx, sizeof(hipfftDoubleComplex) * Nx);
    hipMalloc(&odata_cx, sizeof(hipfftDoubleComplex) * Nx);

    hipMemcpy(idata_c, idata, sizeof(double) * Nx, hipMemcpyHostToDevice);

    /* Convert data into hipfftDoubleComplex */
    /* set 1 block with 256 threads */
    real2complex<<<1, 8>>>(idata_c, idata_cx, Nx);
    hipDeviceSynchronize();

    /* FFT Plans */
    hipfftHandle plan;
    hipfftPlan1d(&plan, Nx, HIPFFT_Z2Z, 1);


    // auto start = chrono::high_resolution_clock::now();
    /* Forward FFT */
    hipEventRecord(start);
    hipfftExecZ2Z(plan, idata_cx, odata_cx, HIPFFT_FORWARD);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    /* stop the time */
    /* std::chrono::_V2::system_clock::time_point finish */ 
    float duration = 0; // milliseconds
    hipEventElapsedTime(&duration, start, stop);
    /* Convert cufft back to double array */
    /* set 1 block with 8 threads */

    complex2real<<<1, 8>>>(odata_cx, odata_c, Nx);
    hipDeviceSynchronize();

    hipMemcpy(odata, odata_c, sizeof(double)*Nx, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(idata_c);
    hipFree(idata_cx);
    hipFree(odata_c);

    return duration;
}

/* convert a double array to cuffComplex data type. Imaginary parts are
 * set to 0 
 */
__global__ void real2complex(double *f, hipfftDoubleComplex *fc, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < N) {
        fc[index].x = f[index];
        fc[index].y = 0;
        index += stride;
    }
}

/* convert a cuffComplex data type to a double array.
 */
 __global__ void complex2real(hipfftDoubleComplex *fc, double *f, int N) {
    /* Assume 1D grid of 1D blocks */
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    while (index < N) {
        f[index] = fc[index].x;
        index += stride;
    }
}